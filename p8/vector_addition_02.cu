#define N 10000000000


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, long int n) {
    for(long int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
	hipError_t cudaerr;

    printf("%d\n", sizeof(long int));

    // Allocate memory in host memory (CPU)
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(long int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

	// Allocate memory in device memory (GPU)
	float *d_a, *d_b, *d_out;
    cudaerr = hipMalloc((void **) &d_a, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_b, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_out, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);

	cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);


}
