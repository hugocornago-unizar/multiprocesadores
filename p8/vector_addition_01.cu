#define N 10000000000


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, long int n) {
    for(long int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(long int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);

	hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

}
