#define N 1024*1024*1024 


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, long int n) {
    long int tid = blockIdx.x * blockDim.x + threadIdx.x; 

	out[tid] = a[tid] + b[tid];
}

int main(){
    float *a, *b, *out; 
	hipError_t cudaerr;

    // Allocate memory in host memory (CPU)
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(long int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

	// Allocate memory in device memory (GPU)
	float *d_a, *d_b, *d_out;
    cudaerr = hipMalloc((void **) &d_a, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_b, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_out, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	// Copy source data from host to device
	cudaerr = hipMemcpy (d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess)
		printf("Copying data host to device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	
	cudaerr = hipMemcpy (d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess)
		printf("Copying data host to device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
	// Main function
    long int block_size = 512;
    long int grid_size = (N / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

	// Copy result data from device to host 
	cudaerr = hipMemcpy (out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess)
		printf("Copying data Device to host failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	//Free memory in the device	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);


}
